#include "hip/hip_runtime.h"
/*

  neurogpu++.cu
  Interface for the neurogpu++ program, that implements MLP Neural Networks
  in CUDA.

  Francisco M. Magalhaes Neto, 2014-05-28
  Based on neurogpu, by Andrei de A. Formiga, 2012-05-21

 */

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

#include "mlpnnets.h"
#include "stopwatch.h"

#define SEED                  631814ULL
#define MAX_ABS               1.2f

DataSet* readDataset(char *filename, int inputSize, int outputSize, bool isFunction)
{
	FILE    *f;
	int     i;
	char    buffer[500];
	char    *value;
	float   outputValue;
	float   *outputs;
	DataSet *dset;

	f = fopen(filename, "r");
	if (f == NULL) {
		fprintf(stderr, "File not found: %s\n", filename);
		return NULL;
	}

	// count lines in file to allocate dataset arrays
	i = 0;
	while (fgets(buffer, 500, f) != NULL)
		++i;

	if (!feof(f) || ferror(f)) {
		fprintf(stderr, "IO error while reading from file\n");
		fclose(f);
		return NULL;
	}
	fseek(f, 0, SEEK_SET);

	dset = CreateDataSet(i, inputSize, outputSize);

	if (dset == NULL) {
		fprintf(stderr, "Error creating dataset\n");
		return NULL;
	}

	int  iix = 0, oix = 0;
	outputs = (float*) malloc(sizeof(float) * outputSize);
	while(fgets(buffer, 500, f) != NULL){

		value = strtok(buffer, ",");
		for (int i = 0; i < inputSize; ++i) {

			dset->inputs[iix++] = atof(value);
			value = strtok(NULL, ",");
		}
		outputValue = atof(value);
		if (outputSize > 1)
			for (int i = 0; i < outputSize; ++i)
				dset->outputs[oix++] = (i == outputValue) ? 0.9f : 0.1f;
		else if (!isFunction)
			dset->outputs[oix++] = (outputValue == 1) ? 0.9f : 0.1f;
		else
			dset->outputs[oix++] = (outputValue + 1.0f) / 2.0f;
	}

	free(outputs);
	fclose(f);

	return dset;
}

void print_dataset(DataSet *dset)
{
	int i, j;

	printf("Number of cases: %d\n", dset->nCases);
	for (i = 0; i < dset->nCases; ++i) {
		for (j = 0; j < dset->inputSize; ++j)
			printf("%3.2f ", dset->inputs[i*dset->inputSize+j]);
		printf(" | ");
		for (j = 0; j < dset->outputSize; ++j)
			printf("%3.2f ", dset->outputs[i*dset->outputSize+j]);
		printf("\n");
	}
}

int outputToClass(float *output, int outputSize)
{
	int classNumber = 0;

	if (outputSize == 1) {
		if (output[0] < 0.5f)
			return 0;

		return 1;
	}

	for (int i = 0; i < outputSize; ++i) {
		if (output[i] > output[classNumber])
			classNumber = i;
	}
	return classNumber;
}

void print_network_data(MLPNetwork *net)
{
	printf("nLayers = %d, d_weights = %lu, nWeights = %d, nCases = %d\n",
	       net->nLayers, (unsigned long) net->d_weights, net->nWeights, net->nCases);
	printf("output ptr for first layer: %lu\n", (unsigned long) net->layers[0]->d_outs);
	printf("output ptr for last layer: %lu\n", (unsigned long) net->layers[net->nLayers-1]->d_outs);
}

struct TestDescription {
	int inputSize;
	int outputSize;
	int epochs;
	float learningRate;
	bool isFunction;
	int nLayers;
	int *neuronsPerLayer;
};

TestDescription *readDescription(const char *name) {
	FILE *f;
	TestDescription *desc;
	char buf[50];
	char *layer;
	int isFunction;

	f = fopen(name, "r");
	if (f == NULL) {
		fprintf(stderr, "File not found: %s\n", name);
		return NULL;
	}
	desc = (TestDescription*) malloc(sizeof(TestDescription));

	fscanf(f, "%d\n", &desc->inputSize);
	fscanf(f, "%d\n", &desc->outputSize);
	fscanf(f, "%d\n", &desc->epochs);
	fscanf(f, "%f\n", &desc->learningRate);

	fscanf(f, "%d\n", &isFunction);
	desc->isFunction = isFunction;

	fscanf(f, "%d\n", &desc->nLayers);

	desc->neuronsPerLayer = (int*) malloc(sizeof(int) * desc->nLayers);

	fgets(buf, 50, f);
	layer = strtok(buf, ",");
	for (int i = 0; i < desc->nLayers; ++i) {
		desc->neuronsPerLayer[i] = atoi(layer);
		layer = strtok(NULL, ",");
	}

	return desc;
}

void destroyDescription(TestDescription *desc) {
	free(desc->neuronsPerLayer);
	free(desc);
}

int runTest(const char *name, int casesPerBlock, int neuronsPerThread) {
	int     i;
	int     errors;
	DataSet *train_set;
	DataSet *test_set;
	TestDescription *desc;
	float   e;
	double  acc;
	StopWatch timer;
	double elapsedTime;
	char desc_name[FILENAME_MAX];
	char train_name[FILENAME_MAX];
	char test_name[FILENAME_MAX];

	MLPNetwork *nn;

	sprintf(desc_name, "data/%s.desc", name);
	sprintf(train_name, "data/%s.train", name);
	sprintf(test_name, "data/%s.test", name);

	desc = readDescription(desc_name);

	// training
	train_set = readDataset(train_name, desc->inputSize, desc->outputSize, desc->isFunction);

	if (train_set == NULL) {
		fprintf(stderr, "Error reading training set\n");
		exit(1);
	}

	nn = CreateNetwork(desc->nLayers, desc->neuronsPerLayer);
	RandomWeights(nn, MAX_ABS, SEED);

	printf("Training network with %d epochs...\n", desc->epochs);
	StartTimer(&timer);
	e = BatchTrainBackprop(nn, train_set, desc->epochs, desc->learningRate,
			true, false, ACTF_SIGMOID, casesPerBlock, neuronsPerThread);
	StopTimer(&timer);
	elapsedTime = GetElapsedTime(&timer);
	printf("Training finished, approximate final MSE: %f\n", e/nn->nCases);

	printf("Weights after training:\n");
	PrintWeights(nn);

	printf("-----------------------------------------\n");

	// free the training dataset
	hipDeviceSynchronize();
	DestroyDataSet(train_set);

	// testing
	test_set = readDataset(test_name, desc->inputSize, desc->outputSize, desc->isFunction);

	if (test_set == NULL) {
		fprintf(stderr, "Error reading test set\n");
		return -1;
	}

	errors = 0;

	if (!PrepareForTesting(nn, test_set->nCases)) {
		fprintf(stderr, "Error preparing network for testing\n");
		return -1;
	}

	printf("Testing with %d cases...\n", test_set->nCases);
	PresentInputsFromDataSet(nn, test_set, ACTF_SIGMOID, 1, 1);

	hipDeviceSynchronize();

	printf("Weights again:\n");
	PrintWeights(nn);

	float *output = (float*) malloc(sizeof(float) * test_set->nCases * test_set->outputSize);

	if (output == NULL) {
		fprintf(stderr, "Could not allocate memory for copying output to host\n");
		return -1;
	}

	if (!CopyNetworkOutputs(nn, output)) {
		fprintf(stderr, "Could not get device outputs\n");
		return -1;
	}

	if (!desc->isFunction) {
		int predicted, desired;
		for (i = 0; i < test_set->nCases; ++i) {
			predicted = outputToClass(output + (i * desc->outputSize), desc->outputSize);
			desired = outputToClass(test_set->outputs + (i * desc->outputSize), desc->outputSize);
			if (predicted != desired)
				++errors;
			printf("Case %d | predicted: %d, desired: %d\n", i,
				   predicted, desired);
		}

		acc = 100.0 - (100.0 * errors / test_set->nCases);
		printf("Testing accuracy: %f\n", acc);
		printf("Total classification errors: %d\n", errors);
	} else {
		float predicted, desired;
		double error = 0.0f;
		for (i = 0; i < test_set->nCases; ++i) {
			predicted = output[i];
			desired = test_set->outputs[i];
			error += fabs(predicted - desired);
			printf("Input %f | predicted: %f, desired: %f\n", test_set->inputs[i],
			       predicted, desired);
		}
		error /= test_set->nCases;
		printf("Mean error: %f\n", error);
	}

	free(output);
	DestroyNetwork(nn);
	DestroyDataSet(test_set);
	destroyDescription(desc);

	fprintf(stderr, "Training time: %lf seconds\n", elapsedTime);

	return 0;
}

int main(int argc, char **argv)
{
	int casesPerBlock = 1;
	int neuronsPerThread = 1;
	char name[FILENAME_MAX] = "cancer";

	if (argc > 1) {
		strcpy(name, argv[1]);
		if (argc == 4) {
			casesPerBlock = atoi(argv[2]);
			neuronsPerThread = atoi(argv[3]);
		} else if (argc != 2) {
			fprintf(stderr, "Syntax error\n");
			exit(EXIT_FAILURE);
		}
	}

	if ((casesPerBlock > 0) && (neuronsPerThread > 0))
		return runTest(name, casesPerBlock, neuronsPerThread);
	fprintf(stderr, "Syntax error\n");
	return EXIT_FAILURE;
}
