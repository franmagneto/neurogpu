#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#define MIN(a, b) (a < b) ? a : b

__global__ void inputs_gen(float *in, int samples, float first, float last) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	float precision = (last - first) / (samples-1);

	if (tid < samples) {
		in[tid] = (tid*precision + first);
	}
}

__global__ void sin_compute(float *in, float *out, int samples) {
	int tid = blockIdx.x * blockDim.x + threadIdx.x;

	if (tid < samples) {
		out[tid] = sinf(in[tid]);
	}
}

int main(int argc, char **argv) {

	float *in, *out;
	float *d_in, *d_out;
	int samples = 12501;
	char option;
	hipError_t err;

	if (argc == 2) {
		option = argv[1][0];
		switch (option) {
			case 'a':
				samples = 12500001;
				break;
			case 'b':
				samples = 1250001;
				break;
			case 'c':
				samples = 125001;
				break;
			case 'd':
				samples = 12501;
				break;
			case 'e':
			default:
				samples = 1251;
		}
	}
	int size = sizeof(float)*samples;

	in = (float*)malloc(size);
	out = (float*)malloc(size);

	if (!in || !out) {
		fprintf(stderr, "Erro alocando vetores\n");
		exit(EXIT_FAILURE);
	}

	err = hipMalloc(&d_in, size);
	if (err != hipSuccess) {
		fprintf(stderr, "Erro alocando entradas (%s)\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	err = hipMalloc(&d_out, size);
	if (err != hipSuccess) {
		fprintf(stderr, "Erro alocando saidas (%s)\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	int tpb = MIN(samples, 512);
    int bpg = (samples-1)/tpb+1;
	inputs_gen<<<bpg, tpb>>>(d_in, samples, -2.0 * M_PI, 2 * M_PI);
	if (hipGetLastError() != hipSuccess) {
		fprintf(stderr, "Erro ao gerar entradas\n");
		exit(EXIT_FAILURE);
	}
	hipDeviceSynchronize();

	sin_compute<<<bpg, tpb>>>(d_in, d_out, samples);
	if (hipGetLastError() != hipSuccess) {
		fprintf(stderr, "Erro ao computar seno\n");
		exit(EXIT_FAILURE);
	}
	hipDeviceSynchronize();

	err = hipMemcpy(in, d_in, size, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		fprintf(stderr, "Erro ao transferir valores de entrada (%s)\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	err = hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost);
	if (err != hipSuccess) {
		fprintf(stderr, "Erro ao transferir valores de saida (%s)\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	hipFree(d_in);
	hipFree(d_out);

	for (int i = 0; i < samples; ++i) {
		printf("%f,%f\n", in[i], out[i]);
	}
	free(in);
	free(out);

	return EXIT_SUCCESS;
}
